/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.cuh>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
using namespace std;

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	int *greyMap;
	int *greyMap2;
	int *filterMap;
	int *filterMap2;
	int *convolutionMask;

	//cudaTest();
	cout << "Hola Mundo" << endl;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	//cout << "RPM " << posicion << endl;
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];
	greyMap = new int[f * c];
	filterMap = new int[f * c];
	greyMap2 = new int[f * c];
	filterMap2 = new int[f * c];

	convolutionMask = new int[9];
	horizontalEdgesMask(convolutionMask);
	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	Image imagen("P3", 255, f, c, map);
	//imagen.convertToGrey();
	//greyMap = imagen.getBitMap();
	cudaConvertToGreyMap(map, greyMap, (f * c));
	verticalEdgesMask(convolutionMask);
	cudaConvolution(greyMap, filterMap, convolutionMask, f, c, 1);
	horizontalEdgesMask(convolutionMask);
	cudaConvolution(greyMap, greyMap2, convolutionMask, f, c, 1);

	testSobel(filterMap, greyMap2, filterMap2, (f*c));
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;

	for (int i = 0; i < f * c; i++) {
		cout << filterMap2[i] << endl;
	}
	//

	//imagen.printImagetoConsole();

	//cout << "RPM alto" << f << " ancho " << c << endl;

	return 0;
}
