#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.h>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
#include <filterApplication.h>

using namespace std;

void testCode(float * v) {
	for(int i=0;i<9;i++){
		cout<<v[i]<< " ";
	}
}

void testCodeStaticSobel(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float *mask = new float[9];

	horizontalEdgesMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	staticConvolution(grayMap, mask, cGrayMap1, 1, f, c);
	verticalEdgesMask(mask);
	staticConvolution(grayMap, mask, cGrayMap2, 1, f, c);
	sumMatrix(cGrayMap1, cGrayMap2, conc, f, c);
}

void testCodeStaticSharpen(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float *mask = new float[9];

	sharpenMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	staticConvolution(grayMap, mask, cGrayMap1, 1, f, c);
}

void testCodeStaticBlur(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float *mask = new float[9];

	blurMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	staticConvolution(grayMap, mask, cGrayMap1, 9, f, c);
}

void testCodeParalellSobel(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float * mask = new float[9];

	horizontalEdgesMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	cudaSobelFilter(grayMap, conc, f, c, 1);
}

void testCodeParalellSharpen(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float * mask = new float[9];

	sharpenMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	cudaConvolution(grayMap, conc, mask, f, c, 1);
}

void testCodeParalellBlur(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	float * mask = new float[9];

	blurMask(mask);
	cudaConvertToGreyMap(image.getBitMap(), grayMap, image.getBipMapLength());
	cudaConvolution(grayMap, conc, mask, f, c, 9);
}

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	float *convolutionKernel;
	int factor = 1;
	vector<ProcessNode> nodes;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}
	if (argv[2]) {
		factor = atoi(argv[2]);
	} else {
		cout << "sin factor " << endl << endl;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];
	convolutionKernel = new float[9];

	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	readConvolutionKernel(convolutionKernel);
	Image imagen("P3", 255, f, c, map);

	if (argv[3] && strcmp(argv[3], "debug") == 0) {
		if (strcmp(argv[4], "ss") == 0) {
			testCodeStaticSobel(imagen);
		} else if (strcmp(argv[4], "ps") == 0) {
			testCodeParalellSobel(imagen);
		} else if(strcmp(argv[4], "ssh") == 0) {
			testCodeStaticSharpen(imagen);
		} else if(strcmp(argv[4], "sb") == 0) {
			testCodeStaticBlur(imagen);
		} else if (strcmp(argv[4], "psh") == 0) {
			testCodeParalellSharpen(imagen);
		}
		else if(strcmp(argv[4], "pb") == 0) {
			testCodeParalellBlur(imagen);
		}
		else {
			cout << "test code " << endl;
			testCode(convolutionKernel);
		}
		return 0;
	}

	if (setEmptyPipeline(&nodes, imagen)) {
		pipelineIterator(&nodes, imagen, factor);
	} else {
		cout << "No se pudo instanciar un pipeline vacio" << endl;
		return 0;
	}
	int size = 0;
	size = nodes.size() - 1;
	writeGpmImage(nodes[size].getOutputImage().getBitMap(), f, c, maximo);
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	return 0;

}
