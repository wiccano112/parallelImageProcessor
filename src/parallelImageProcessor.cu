/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.cuh>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
using namespace std;

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	int *greyMap;
	int *greyMap2;
	int *filterMap;
	int *filterMap2;
	int *convolutionMask;

	//cudaTest();
	cout << "Hola Mundo" << endl;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	//cout << "RPM " << posicion << endl;
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];
	greyMap = new int[f * c];
	filterMap = new int[f * c];
	greyMap2 = new int[f * c];
	filterMap2 = new int[f * c];

	//convolutionMask = new int[9];
	//horizontalEdgesMask(convolutionMask);
	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	Image imagen("P3", 255, f, c, map);
	//imagen.convertToGrey();
	//greyMap = imagen.getBitMap();
	cudaConvertToGreyMap(map, greyMap, (f * c));
	int opcion = 1;
	int convolucionNumber = 1;
	while (opcion) {
		cudaSobelFilter(greyMap, filterMap, f, c, convolucionNumber);
		writeGpmImage(filterMap, f, c, maximo);
		cout << "repetimos?:(0 | 1) ";
		cin >> opcion;
		if(!opcion){
			break;
		}
		cout << "ingresar nuevo valor de convolucion: ";
		cin >> convolucionNumber;

	}
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	//

	//imagen.printImagetoConsole();

	//cout << "RPM alto" << f << " ancho " << c << endl;

	return 0;
}
