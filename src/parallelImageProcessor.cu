#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Image.h>
#include <filterApplication.h>
#include <profilingMethods.h>
#include <Util.h>

using namespace std;

/*
 * variables globales
 */
bool debug=false;

void testCode(float * v) {
	for(int i=0;i<9;i++){
		cout<<v[i]<< " ";
	}
}

int main(int argc, char **argv) {

	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	float *convolutionKernel;
	int factor = 1;
	vector<ProcessNode> nodes;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}

	//readInput(argc, argv);

	if (argv[2]) {
		factor = atoi(argv[2]);
	} else {
		cout << "sin factor " << endl << endl;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];
	convolutionKernel = new float[9];

	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	readConvolutionKernel(convolutionKernel);
	Image imagen("P3", 255, f, c, map);

	if (argv[3] && strcmp(argv[3], "debug") == 0) {
		if (strcmp(argv[4], "ss") == 0) {
			testCodeStaticSobel(imagen);
		} else if (strcmp(argv[4], "ps") == 0) {
			testCodeParalellSobel(imagen);
		} else if(strcmp(argv[4], "ssh") == 0) {
			testCodeStaticSharpen(imagen);
		} else if(strcmp(argv[4], "sb") == 0) {
			testCodeStaticBlur(imagen);
		} else if (strcmp(argv[4], "psh") == 0) {
			testCodeParalellSharpen(imagen);
		}
		else if(strcmp(argv[4], "pb") == 0) {
			testCodeParalellBlur(imagen);
		}
		else {
			cout << "test code " << endl;
			testCode(convolutionKernel);
		}
		return 0;
	}

	if (setEmptyPipeline(&nodes, imagen)) {
		pipelineIterator(&nodes, imagen, factor);
	} else {
		cout << "No se pudo instanciar un pipeline vacio" << endl;
		return 0;
	}
	int size = 0;
	size = nodes.size() - 1;
	writeGpmImage(nodes[size].getOutputImage().getBitMap(), f, c, maximo);
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	return 0;

}
