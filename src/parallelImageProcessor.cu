/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.h>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
#include <filterApplication.h>

using namespace std;

void testCode(vector<ProcessNode> *nodes, Image image) {

	//doing iterator for pipeline
	int actual = 0;

	for (int i = actual; i < nodes->size(); i++) {
		//cout << "RPM numero de filtro " << nodes[0][i].getFilter() << endl;
		if (i == 0) {
			//initial node
			nodes[0][i + 1].setInputImage(nodes[0][i].getInputImage());
		} else if (i == nodes->size() - 1) {
			//last node
			nodes[0][i].setOutputImage(nodes[0][i].getInputImage());
		} else {
			//filter node
			nodes[0][i].setOutputImage(
					doFilter(nodes[0][i].getInputImage(),
							nodes[0][i].getFilter()));
			nodes[0][i + 1].setInputImage(nodes[0][i].getOutputImage());
		}
	}
}

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	vector<ProcessNode> nodes;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];

	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	Image imagen("P3", 255, f, c, map);

	if (setEmptyPipeline(&nodes, imagen)) {
		cout << "cantidad de nodos " << nodes.size() << endl;
		pipelineIterator(&nodes, imagen);
		//TODO doing iterator for pipeline
	} else {
		cout << "todo mal" << endl;
		return 0;
	}
	int size = 0;
	size = nodes.size() - 1;
	writeGpmImage(nodes[size].getOutputImage().getBitMap(), f, c, maximo);
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	return 0;

}
