#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.h>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
#include <filterApplication.h>

using namespace std;

void testCode(vector<ProcessNode> *nodes, Image image) {

}

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	int factor = 1;
	vector<ProcessNode> nodes;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}
	if (argv[2]) {
		factor = atoi(argv[2]);
	} else {
		cout << "sin factor " << endl << endl;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];

	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	Image imagen("P3", 255, f, c, map);

	if (setEmptyPipeline(&nodes, imagen)) {
		pipelineIterator(&nodes, imagen, factor);
	} else {
		cout << "todo mal" << endl;
		return 0;
	}
	int size = 0;
	size = nodes.size() - 1;
	writeGpmImage(nodes[size].getOutputImage().getBitMap(), f, c, maximo);
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	return 0;

}
