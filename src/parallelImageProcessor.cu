#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <cstring>
#include <Util.h>
#include <Image.h>
#include <hip/hip_runtime.h>
#include <cudaSources.cuh>
#include <filterApplication.h>

using namespace std;

void testCodeStaticSobel(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	//cout <<f<<" "<<c<<endl;
	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	int * mask = new int[9];
	int g = 0;
	//while (g != 30) {
		horizontalEdgesMask(mask);
		cudaConvertToGreyMap(image.getBitMap(), grayMap,
				image.getBipMapLength());
		staticSobel(grayMap, mask, cGrayMap1, 1, f, c);
		verticalEdgesMask(mask);
		staticSobel(grayMap, mask, cGrayMap2, 1, f, c);
		sumMatrix(cGrayMap1, cGrayMap2, conc, f, c);
		g++;
	//	}
//	cout << "P2" << endl << "#do for test" << endl << f << " " << c << endl
//			<< "255" << endl;
//	for (int i = 0; i < f * c; i++) {
//		cout << conc[i] << endl;
//	}
}

void testCodeParalellSobel(Image image) {
	int f = image.getLength();
	int c = image.getWidth();

	//cout <<f<<" "<<c<<endl;
	int * grayMap = new int[f * c];
	int * cGrayMap1 = new int[f * c];
	int * cGrayMap2 = new int[f * c];
	int * conc = new int[f * c];
	int * mask = new int[9];
	int g = 0;
	//while (g != 30) {
		horizontalEdgesMask(mask);
		cudaConvertToGreyMap(image.getBitMap(), grayMap,image.getBipMapLength());
		cudaSobelFilter(grayMap,conc,f,c,1);
		g++;
	//}

}

int main(int argc, char **argv) {
	char tamano[20];
	int maximo = 0;
	int posicion = 0;
	int f = 0;
	int c = 0;
	int *map;
	int factor = 1;
	vector<ProcessNode> nodes;

	if (argc <= 1) {
		cout << "no ingresaste archivos por linea de comandos" << endl;
		return 0;
	}
	if (argv[2]) {
		factor = atoi(argv[2]);
	} else {
		cout << "sin factor " << endl << endl;
	}

	posicion = filePreProcess(argv[1], tamano, maximo);
	c = getLengthFromString(tamano);
	f = getWidthFromString(tamano);
	map = new int[f * c * 3];

	bitMapBuilder(posicion, argv[1], f, c, map, 3);
	Image imagen("P3", 255, f, c, map);

	if (argv[3] && strcmp(argv[3], "debug") == 0 ) {
		if(strcmp(argv[4], "s") == 0) {
			testCodeStaticSobel(imagen);
		}
		else if(strcmp(argv[4], "p") == 0) {
			testCodeParalellSobel(imagen);
		}
		return 0;
	}

	if (setEmptyPipeline(&nodes, imagen)) {
		pipelineIterator(&nodes, imagen, factor);
	} else {
		cout << "todo mal" << endl;
		return 0;
	}
	int size = 0;
	size = nodes.size() - 1;
	writeGpmImage(nodes[size].getOutputImage().getBitMap(), f, c, maximo);
	hipError_t err;
	err = hipDeviceSynchronize();
	cout << hipGetErrorString(err) << endl;
	return 0;

}
